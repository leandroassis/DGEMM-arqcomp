// compile com nvcc .\DGEMM.cu -o saida -O3 -Xptxas -O3 -lineinfo -arch=sm_61 -use_fast_math
// verifique a arquitetura da sua GPU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>

#define SEED                 0x12345678

// tamanho da matriz
#define NUM_ITERACOES       5 // numero de iteracoes para cada tamanho de matriz
#define TAMANHO_MIN_MATRIZES 4096 // 4096 linhas e 4096 colunas = 16.777.216 células aprox 16MB
#define TAMANHO_MAX_MATRIZES 8192 // 8192 linhas e 8192 colunas = 67.108.864 células aprox 64MB
#define NUM_RESULTADOS_DESEJADOS 4
#define STEP_MATRIZES        (TAMANHO_MAX_MATRIZES-TAMANHO_MIN_MATRIZES)/NUM_RESULTADOS_DESEJADOS // deve ser multiplo de 32 assim como os tamanhos min e max das matrizes

// valores inseridos na matriz
#define MIN_DOUBLE           -100000.0
#define MAX_DOUBLE            100000.0

#define BLOCK_SIZE 4

double rand_double(){
    return (MIN_DOUBLE + (rand()*(MAX_DOUBLE - MIN_DOUBLE)/RAND_MAX));
}

// Função do kernel CUDA para multiplicação de matrizes com uso de "SIMD" (via paralelização de threads)
__global__ void DGEMM(double* A, double* B, double* C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        double sum = 0.0f;

        for (int k = 0; k < size; k++) {
            sum += A[row * size + k] * B[k * size + col];
        }

        C[row * size + col] = sum;
    }
}


int main() {

    hipEvent_t t_init, t_fim;

    hipEventCreate(&t_init);
    hipEventCreate(&t_fim);

    float t_exe;
    FILE *fp;

    srand(SEED);

    fp = fopen("resultadosCUDA.csv", "w"); // RESULTADOS
    fprintf(fp, "Tamanho da matriz;Tempo DGEMM\n");
    
    for(size_t size = TAMANHO_MIN_MATRIZES; size <= TAMANHO_MAX_MATRIZES; size+=STEP_MATRIZES){

        // Alocação e inicialização das matrizes no host (CPU)
        double* A = (double*) malloc(size*size*sizeof(double));
        double* C = (double*) malloc(size*size*sizeof(double));
        double* B = (double*) malloc(size*size*sizeof(double));

        // Alocação das matrizes no device (GPU)
        size_t matrixSize = size*size*sizeof(double);
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, matrixSize);
        hipMalloc((void**)&d_B, matrixSize);
        hipMalloc((void**)&d_C, matrixSize);

        int i;
        for (i = 0; i < size*size; i++) {
            A[i] = (double) rand_double();
            B[i] = (double) rand_double();
            C[i] = (double) rand_double();
        }

        // Cópia das matrizes da CPU para a GPU
        hipMemcpy(d_A, A, matrixSize, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, matrixSize, hipMemcpyHostToDevice);

        for(int i = 0; i < NUM_ITERACOES; i++){
            fprintf(fp, "%zu;", size*size);

            // Configuração das dimensões do grid e dos blocos
            dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
            dim3 gridSize((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE);

            // Chamada do kernel CUDA para multiplicação de matrizes

            hipEventRecord(t_init); // tempo de inicio
            DGEMM<<<gridSize, blockSize>>>(d_A, d_B, d_C, size);
            hipEventRecord(t_fim);
            
            hipEventSynchronize(t_fim);
            hipEventElapsedTime(&t_exe, t_init, t_fim);

            printf("Tempo de execução do algoritmo DGEMM normal com matriz de tamanho %.2f MB: %f s.\n", (float) (size*size/(1024*1024)), t_exe/1e6);
            fprintf(fp, "%f\n", t_exe);

            // Cópia do resultado da GPU para a CPU
            // nao é necessario copiar o resultado para a CPU
            //cudaMemcpy(C, d_C, matrixSize, cudaMemcpyDeviceToHost);

        }

        // Liberação da memória
        free(A);
        free(B);
        free(C);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    fclose(fp);

    return 0;
}
